#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <chrono>
#include <time.h>
#include <iostream>
#include <cfloat>
#include <algorithm>  // not sure if needed

//for CUDA garbage - maybe useful
#include <hip/hip_runtime.h>


//#define int LENGTH 100     // length of data set - okay this doesnt work aparently

int LENGTH = 10000;


//this is possible error, not save mem correctly
#if __CUDA_ARCH__ < 600  // allows us to use atomicMin with doubles on our old as dirt CUDA versions
__device__ double atomicMinf(double* address, double val){

    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    while (val < __longlong_as_double(old)) {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,  __double_as_longlong(val)); // (old == assumed ? val : old)
    }
    return __longlong_as_double(old);
}
#endif

//CUDA kernal for part one
__global__ void findLeast(const double *array, double *m, const int size){
  extern __shared__ double share[]; // extern : "size determined at runtime by the kernel's caller via a launch configuration argument" - whatever that means
  int tid = threadIdx.x;
  int gid = blockDim.x * blockIdx.x + threadIdx.x; // what element to work on
  share[tid] = DBL_MAX; // initalize with largest num

  while(gid < size){  //check to see if in range
    share[tid] = max(share[tid], array[gid]);
    gid += gridDim.x*blockDim.x;  // what element each thread should work on
  }
  __syncthreads();
  gid = blockDim.x * blockIdx.x + threadIdx.x; // reset gid for future calculations

  // reduce spread out shared memory in block into one location: shared[0]
  for(int i = blockDim.x / 2; i > 0; i/=2){   // note blockDim.x is number of threads in a block, always even
    if(tid < i and gid < size)
      share[tid] = max(share[tid], share[tid + i]);
    __syncthreads();
  }

  // compare across blocks to find true min
  if(tid == 0)
    atomicMinf(m, share[0]);    // or is the error here, no clue
}

///////////////////////////////////////////////////////////////////////////////

void fillArray(double *n, int s){
  std::srand(std::time(NULL));    // lets seed rand with time for fun!!
  for(int i = 0; i < s; i++){
    n[i] =  (double)std::rand();
  }
}

void printArray(float *n, int s){
  for(int i = 0; i < s; i++){
    printf("%.5f ", n[i]);
  }
}

//finish
double checkMin(double* a, int size){
  double min = DBL_MAX;

  return min;
}

int main(int argc, char **argv){

  size_t size = LENGTH * sizeof(double);
  double *h_a = (double*)malloc(size); // allocate mem for host array
  double *output = (double*)malloc(sizeof(double)); // allocate memory for output
  hipError_t err = hipSuccess; // error check, maybe gets implemented

  if(h_a == NULL || output == NULL){
    fprintf(stderr, "Failed to allocate main memory");
    exit(EXIT_FAILURE);
  }
  // fill the array with random values, fill output with max value
  fillArray(h_a, LENGTH);
  *output = DBL_MAX;

  // allocate memory on device for input vector a
  double *d_a = NULL;
  err = hipMalloc(&d_a, size);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  // allocate memory on device for output o
  double *d_o = NULL;
  err = hipMalloc(&d_o, sizeof(double));
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate output (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // copy main memory data into device memory
  err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy vector a from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_o, output, sizeof(double), hipMemcpyHostToDevice);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy output from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ready for takeoff, luanch CUDA kernal
  int threadsPerBlock = 1024;
  int blocksPerGrid = (LENGTH + threadsPerBlock - 1) / threadsPerBlock;
  findLeast<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_o, LENGTH);
  err = hipGetLastError();
  if(err != hipSuccess){
    fprintf(stderr,"(error code %s)\nYou done screwed up\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(&output, d_o, sizeof(double), hipMemcpyDeviceToHost);
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy device d_o to host output (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  //this should be where we verify the output

  printf("The minimum number: %f\n", *output);

  hipFree(d_a);
  hipFree(d_o);
  free(h_a);
  free(output);
  return 0;
}
